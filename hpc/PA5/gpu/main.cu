#include "hip/hip_runtime.h"
/*
    Michael Kmak    
    ECPE 251 - High-Performance Computing
    PA4 - Canny Edge Sate 1 - GPU

    usage: ./canny <image path> <sigma>

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_math_constants.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include "image_template.h"

#define GPU_NO 1 // 85 % 4
#define blocksize 8
#define conv_size 256

//#define debug_mode


void print_k(float *k, int len);
float timecalc(struct timeval start, struct timeval end);
void g_kern(float *k, float sigma);
void g_deriv(float *k, float sigma);


__global__
void gpu_hconvolve(float *img, float *out, int width, int height, float *kern, int kern_w) {
    extern __shared__ float smem[]; 
    float *s_kern = smem;
    float *s_img = &smem[kern_w];
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y; 
    int localidx = threadIdx.x;
    int localidy = threadIdx.y;
    int globalidx = localidx + blockIdx.x*blockDim.x;
    int globalidy = localidy + blockIdx.y*blockDim.y;
    int base = i*width + j;
    int p;

    // load to shared mem
    if (localidy < kern_w)
        s_kern[localidy] = kern[localidy];
    s_img[localidy] = img[globalidx*width + globalidy];
    __syncthreads();

    if (i < height && j < width) {
        float sum = 0;
        for (size_t k = 0; k < kern_w; k++) {
            int offset = k - floorf(kern_w/2);
            p = base + offset;
            if (p / width == base / width) { // same row
                if (p >= 0 && p <= height * width) {
                    if (localidy + offset < blockDim.y)
                        sum += s_img[localidy + offset] * s_kern[k];
                    else
                        sum += img[p] * kern[k];
                }
            }
        }
        out[base] = sum;
    }
}


__global__
void gpu_vconvolve(float *img, float *out, int width, int height, float *kern, int kern_w) {
    extern __shared__ float smem[];
    float *s_kern = smem;
    float *s_img = &smem[kern_w];
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y; 
    int localidx = threadIdx.x;
    int localidy = threadIdx.y;
    int globalidx = localidx + blockIdx.x*blockDim.x;
    int globalidy = localidy + blockIdx.y*blockDim.y;
    int base = i*width + j;
    int p;

    // load to shared mem
    if (localidx < kern_w)
        s_kern[localidx] = kern[localidx];
    s_img[localidx] = img[globalidx*width + globalidy];
    __syncthreads();


    if (i < height && j < width) {
        float sum = 0;
        for (size_t k = 0; k < kern_w; k++) {
            int offset = (k - floorf(kern_w/2)) * width;
            p = base + offset;
            if (p >= 0 && p < width * height) {
                if (localidx + offset < blockDim.x) 
                    sum += s_img[localidx + offset] * s_kern[k];
                else
                    sum += img[p] * s_kern[k];
            }
        }
        out[base] = sum;
    }
}

__global__
void gpu_magdir(float *hori, float *vert, float *mag, float *dir, int height, int width) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = i*width+j;
    float ho = hori[k];
    float ve = vert[k];

    mag[k] = sqrtf((ho * ho) + (ve * ve));
    dir[k] = atan2f(ho, ve);
} 


__global__
void gpu_suppression(float *mag, float *dir, float *supp, int height, int width) {
    extern __shared__ float Gxy[];
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y; 
    int k = j*width + i;
    int local_i = threadIdx.x;
    int local_j = threadIdx.y;
    int l = local_j * blockDim.x + local_i;
    int bounds = width * height;
    int btm_right = width + 1;
    int btm_left = width - 1;
    int l_br = blockDim.x + 1;
    int l_bl = blockDim.x - 1;
    
    // load shared mem
    Gxy[l] = mag[k];

    float homeval = mag[k];
    float theta = dir[k];
    float suppval = mag[k];
    if (theta <= 0)
        theta += HIP_PI_F;
    theta *= (180.0 / HIP_PI_F);
    if (theta <= 22.5 || theta > 157.5) {
        // top
        if (k >= width) {
            if (local_j > 0) {
                if (homeval < Gxy[l - blockDim.x]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k - width]) {
                    suppval = 0;
                }
            }
        }
        // bottom
        if (k < bounds - width) {
            if (local_j < blockDim.y-1) {
                if (homeval < Gxy[l + blockDim.x]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k + width]) {
                    suppval = 0;
                }
            }
        }
    } else if (theta > 22.5 && theta <= 67.5) {
        //topleft
        if (k >= width && k % width > 0) {
            if (local_j > 0 && local_i > 0) {
                if (homeval < Gxy[l - l_br]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k - btm_right]) {
                    suppval = 0;
                }
            }
        }
        // bottomright
        if (k < bounds - width && k % width < width-1) {
            if (local_j < blockDim.y-1 && local_i < blockDim.x-1) {
                if (homeval < Gxy[l + l_br]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k + btm_right]) {
                    suppval = 0;
                }
            }
        }
    } else if (theta > 67.5 && theta <= 112.5) {
        // left
        if (k % width > 0) {
            if (local_i > 0) {
                if (homeval < Gxy[l - 1]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k - 1]) {
                    suppval = 0;
                }
            }
        }       
        // right
        if (k % width < width-1) {
            if (local_i < blockDim.x-1) {
                if (homeval < Gxy[l + 1]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k + 1]) {
                    suppval = 0;
                }
            }
        }
    } else if (theta > 112.5 && theta <= 157.5) {
        // topright
        if (k >= width && k % width < width-1) {
            if (local_j > 0 && local_i < blockDim.x-1) {
                if (homeval < Gxy[l - l_bl]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k - btm_left]) {
                    suppval = 0;
                }
            }
        }
        // bottomleft
        if (k < bounds - width && k % width > 0) {
            if (local_j < blockDim.y-1 && local_i > 0) {
                if (homeval < Gxy[l + l_bl]) {
                    suppval = 0;
                }
            } else {
                if (homeval < mag[k + btm_left]) {
                    suppval = 0;
                }
            }
        }
    }
    supp[k] = suppval;
}


__global__
void gpu_hysteresis(float *hyst, float width, float t_hi, float t_lo) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = i*width+j;

    if (k < width * width) {
        if (hyst[k] >= t_hi) {
            hyst[k] = 255.0;
        } else if (hyst[k] <= t_lo) {
            hyst[k] = 0.0;
        } else {
            hyst[k] = 125.0;
        }
    }
}


__global__
void gpu_edgelinking(float *hyst, float *edge, int width, int height) { 
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = i*width+j;
    int bounds = width * height;
    int btm_right = width + 1;
    int btm_left = width - 1;

    if(hyst[k] == 125) {
        int edgeval = 0;
        // topleft
        if (k >= width && k % width > 0) {
            if (hyst[k - btm_right] == 255) {
                edgeval = 255;
            }
        }
        // top
        if (k >= width) {
            if (hyst[k - width] == 255) {
                edgeval = 255;
            }
        }
        // topright
        if (k >= width && k % width < width-1) {
            if (hyst[k - btm_left] == 255) {
                edgeval = 255;
            }
        }
        // left
        if (k % width > 0) {
            if (hyst[k - 1] == 255) {
                edgeval = 255;
            }
        }
        // right
        if (k % width > width-1) {
            if (hyst[k + 1] == 255) {
                edgeval = 255;
            }
        }
        // bottomleft
        if (k < bounds - width && k % width > 0) {
            if (hyst[k + btm_left] == 255) {
                edgeval = 255;
            }
        }
        // bottom
        if (k < bounds - width) {
            if (hyst[k + width] == 255) {
                edgeval = 255;
            }
        }
        // bottomright
        if (k < bounds - width && k % width < width-1) {
            if (hyst[k + btm_right] == 255) {
                edgeval = 255;
            }
        }
        edge[k] = edgeval;
    } else {
        edge[k] = hyst[k];
    }
}


int main(int argc, char *argv[]) {

    int height;
    int width;
    float sigma;
    int kern_w;
    struct timeval compstart, compend;
    #ifdef debug_mode
    struct timeval start, stop;
    float commtime, convtime, magdirtime, supptime, sorttime, hysttime, edgetime;
    #endif

    // host
    float *h_img;
    float *h_mag;
    float *h_dir;
    float *h_supp;
    float *h_hyst;
    float *h_edge;
    float *h_vkern;
    float *h_hkern;
    float *h_vderiv;
    float *h_hderiv;

    // device
    float *d_img;
    float *d_temp;
    float *d_hori;
    float *d_vert;
    float *d_mag;
    float *d_dir;
    float *d_supp;
    float *d_hyst;
    float *d_edge;
    float *d_vkern;
    float *d_hkern;
    float *d_vderiv;
    float *d_hderiv;
 
    // argparse
    if (argc != 3) {
        fprintf(stderr, "usage: ./canny <image path> <sigma>\n");
        return -1;
    }
    sigma = atof(argv[2]);
    if (sigma <= 0) {
        fprintf(stderr, "invalid sigma: %s\n", argv[2]);
        return -1;
    }

    // cuda setup
    hipSetDevice(GPU_NO);

    // image prep
    read_image_template<float>(argv[1], &h_img, &width, &height);
    h_mag = (float *) calloc(width*height, sizeof(float));
    h_dir = (float *) calloc(width*height, sizeof(float));
    h_supp = (float *) calloc(width*height, sizeof(float));
    h_hyst = (float *) calloc(width*height, sizeof(float));
    h_edge = (float *) calloc(width*height, sizeof(float));
    hipMalloc((void **)&d_img, sizeof(float)*width*height);
    hipMalloc((void **)&d_temp, sizeof(float)*width*height);
    hipMalloc((void **)&d_hori, sizeof(float)*width*height);
    hipMalloc((void **)&d_vert, sizeof(float)*width*height);
    hipMalloc((void **)&d_mag, sizeof(float)*width*height);
    hipMalloc((void **)&d_dir, sizeof(float)*width*height);
    hipMalloc((void **)&d_supp, sizeof(float)*width*height);
    hipMalloc((void **)&d_hyst, sizeof(float)*width*height);
    hipMalloc((void **)&d_edge, sizeof(float)*width*height);

    // computation start
    gettimeofday(&compstart, NULL);

    // prepare canny kernels
    kern_w = 2 * round(2.5 * sigma - 0.5) + 1;
    h_vkern = (float *) calloc(kern_w, sizeof(float));
    h_hkern = (float *) calloc(kern_w, sizeof(float));
    h_vderiv = (float *) calloc(kern_w, sizeof(float));
    h_hderiv = (float *) calloc(kern_w, sizeof(float));
    hipMalloc((void **)&d_vkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_vderiv, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hderiv, sizeof(float)*kern_w);

    // calculate ckernels
    g_kern(h_vkern, sigma);
    g_kern(h_hkern, sigma);
    g_deriv(h_vderiv, sigma);
    g_deriv(h_hderiv, sigma);

    // transfer ckernels
    #ifdef debug_mode
    gettimeofday(&start, NULL);
    #endif
    hipMemcpy(d_vkern, h_vkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hkern, h_hkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_vderiv, h_vderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hderiv, h_hderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);

    // transfer image
    hipMemcpy(d_img, h_img, sizeof(float)*width*height, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    commtime = timecalc(start, stop);
    #endif

    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&start, NULL);
    #endif

    // horizontal convolve
    dim3 h_dB(1, conv_size);
    dim3 h_dG(height, width/conv_size);
    int memsize = sizeof(float) * (conv_size + kern_w);
    gpu_hconvolve<<<h_dG,h_dB,memsize>>>(d_img, d_temp, width, height, d_hkern, kern_w);
    gpu_hconvolve<<<h_dG,h_dB,memsize>>>(d_temp, d_hori, width, height, d_hderiv, kern_w);

    // vertical convolve
    dim3 v_dB(conv_size, 1);
    dim3 v_dG(height/conv_size, width);
    gpu_vconvolve<<<v_dG,v_dB,memsize>>>(d_img, d_temp, width, height, d_vkern, kern_w);
    gpu_vconvolve<<<v_dG,v_dB,memsize>>>(d_temp, d_vert, width, height, d_vderiv, kern_w);
    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    convtime = timecalc(start, stop);
    gettimeofday(&start, NULL);
    #endif

    //mag & dir
    dim3 dB(blocksize, blocksize);
    dim3 dG(width/blocksize, height/blocksize);
    gpu_magdir<<<dG,dB>>>(d_hori, d_vert, d_mag, d_dir, height, width);
    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    magdirtime = timecalc(start, stop);
    gettimeofday(&start, NULL);
    #endif

    // suppression
    memsize = sizeof(float)*blocksize*blocksize;
    gpu_suppression<<<dG,dB,memsize>>>(d_mag, d_dir, d_supp, height, width);
    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    supptime = timecalc(start, stop);
    gettimeofday(&start, NULL);
    #endif

    // sorting
    hipMemcpy(d_hyst, d_supp, sizeof(float)*height*width, hipMemcpyDeviceToDevice);
    thrust::device_ptr<float> thr_d(d_hyst);
    thrust::device_vector<float> d_hyst_vec(thr_d,thr_d+(height*width));
    thrust::sort(d_hyst_vec.begin(),d_hyst_vec.end());
    int index = (int) (0.9 * height*width);
    float t_hi = d_hyst_vec[index];
    float t_lo = t_hi * 0.2;
    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    sorttime = timecalc(start, stop);
    gettimeofday(&start, NULL);
    #endif


    // hysteresis
    gpu_hysteresis<<<dG,dB>>>(d_hyst, width, t_hi, t_lo);
    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    hysttime = timecalc(start, stop);
    gettimeofday(&start, NULL);
    #endif


    // edge linking
    gpu_edgelinking<<<dG,dB>>>(d_hyst, d_edge, height, width);
    hipDeviceSynchronize();
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    edgetime = timecalc(start, stop);
    gettimeofday(&start, NULL);
    #endif

    // pull results
    #ifdef debug_mode 
    hipMemcpy(h_mag, d_mag, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipMemcpy(h_dir, d_dir, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipMemcpy(h_supp, d_supp, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipMemcpy(h_hyst, d_hyst, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    #endif
    hipMemcpy(h_img, d_edge, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 
    #ifdef debug_mode
    gettimeofday(&stop, NULL);
    commtime += timecalc(start, stop);
    #endif

    // computation end
    gettimeofday(&compend, NULL);

    // write results
    #ifdef debug_mode
    write_image_template<float>("magnitude.pgm", h_mag, width, height);
    write_image_template<float>("direction.pgm", h_dir, width, height);
    write_image_template<float>("suppression.pgm", h_supp, width, height);
    write_image_template<float>("hysteresis.pgm", h_hyst, width, height);
    #endif
    write_image_template<float>("out.pgm", h_img, width, height);

    #ifndef debug_mode
    printf("%d, %0.3f\n", height, timecalc(compstart, compend)); 
    #else
    //printf("idx=%d, hi=%0.2f, lo=%0.2f\n", index, t_hi, t_lo);
    printf("%0.3f, %0.3f, %0.3f, %0.3f, %0.3f, %0.3f, %0.3f, %0.3f\n",
        convtime,
        magdirtime,
        supptime,
        sorttime,
        hysttime,
        edgetime,
        commtime,
        timecalc(compstart, compend)
    );
    #endif

    // free
    free(h_vkern);
    free(h_hkern);
    free(h_vderiv);
    free(h_hderiv);
    free(h_mag);
    free(h_dir);
    free(h_supp);
    free(h_hyst);
    free(h_edge);
    hipFree(d_img);
    hipFree(d_temp);
    hipFree(d_hori);
    hipFree(d_vert);
    hipFree(d_mag);
    hipFree(d_dir);
    hipFree(d_supp);
    hipFree(d_hyst);
    hipFree(d_edge);
}


void print_k(float *k, int len) {
    for (size_t i = 0; i < len; i++) {
        printf("[%f]", k[i]);
    }
    printf("\n");
}


float timecalc(struct timeval start, struct timeval end) {
    float ns = (end.tv_sec*1000000 + end.tv_usec) - (start.tv_sec*1000000 + start.tv_usec);
    return ns / 1000.0;
}


void g_kern(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;

    for (size_t i = 0; i < w; i++) {
        k[i] = exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum += k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
}


void g_deriv(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;
    
    for (size_t i = 0; i < w; i++) {
        k[i] = -1.0 * (i-a) * exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum -= i * k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
    // flip
    for (size_t i = 0; i < (w/2); i++) {
        float temp = k[w-1-i];
        k[w-1-i] = k[i];
        k[i] = temp;
    }
}
