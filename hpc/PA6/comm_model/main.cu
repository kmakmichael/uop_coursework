/*
    Michael Kmak
    ECPE 251 - High-Performance Computing
    PA6 - Performance Prediction

    usage: ./memcpy size

    copies a randomly-filled float array between
    the CPU and GPU in both directions. size of
    this array is 2^n floats, where n is given as
    a program argument. outputs time taken in 
    the following format:
        size H2D D2H
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>

#define GPU_NO 1 // 85 % 4

float timecalc(struct timeval start, struct timeval end) {
    float us = (end.tv_sec*1000000 + end.tv_usec) - (start.tv_sec*1000000 + start.tv_usec);
    return us / 1000.0;
}

int main(int argc, char *argv[]) {
    
    float *h_arr;
    float *d_arr;
    size_t size;
    size_t n;

    struct timeval start, end;
    float htod, dtoh;

    // argparse
    if (argc != 2) {
        fprintf(stderr, "usage: ./memcpy <size>\n");
        return 1;
    }
    int success = sscanf(argv[1], "%zu", &n);
    if (success != 1) {
        fprintf(stderr, "invalid size, enter an integer\n");
        return -1;
    }
    if (n > log2((float)SIZE_MAX)) {
        fprintf(stderr, "enter a power of two no larger than %f\n", log2((float)SIZE_MAX));
        return -1;
    }

    size = exp2((float)n) * sizeof(float);
    h_arr = (float *) malloc(size);
    hipMalloc((void **)&d_arr, size);

    hipDeviceSynchronize();
    gettimeofday(&start, NULL);
    hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice); 
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    htod = timecalc(start, end);

    hipDeviceSynchronize();
    gettimeofday(&start, NULL);
    hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    dtoh = timecalc(start, end);

    char sizeout[8];
    switch (n/10) {
        case 0:
            sprintf(sizeout, "%0.0f", exp2((float)(n%10)));
            break;
        case 1:
            sprintf(sizeout, "%0.0fK", exp2((float)(n%10)));
            break;
        case 2:
            sprintf(sizeout, "%0.0fM", exp2((float)(n%10)));
            break;
        case 3:
            sprintf(sizeout, "%0.0fG", exp2((float)(n%10)));
            break;
        default:
            sprintf(sizeout, "err");
    }
    printf("%s, %0.2f, %0.2f\n", sizeout, htod, dtoh);

    free(h_arr);
    hipFree(d_arr);
}
