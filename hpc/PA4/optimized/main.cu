#include "hip/hip_runtime.h"
/*
    Michael Kmak    
    ECPE 251 - High-Performance Computing
    PA4 - Canny Edge Sate 1 - GPU

    usage: ./canny <image path> <sigma>

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

//#include "sort.h"
#include "image_template.h"

#define GPU_NO 1 // 85 % 4
#define blocksize 8
#define conv_size 512


void print_k(float *k, int len);
float timecalc(struct timeval start, struct timeval end);
void g_kern(float *k, float sigma);
void g_deriv(float *k, float sigma);


__global__
void gpu_hconvolve(float *img, float *out, int width, int height, float *kern, int kern_w) {
    extern __shared__ float smem[]; 
    float *s_kern = smem;
    float *s_img = &smem[kern_w];
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y; 
    int localidx = threadIdx.x;
    int localidy = threadIdx.y;
    int globalidx = localidx + blockIdx.x*blockDim.x;
    int globalidy = localidy + blockIdx.y*blockDim.y;
    int base = i*width + j;
    int p;

    // load to shared mem
    if (localidy < kern_w)
        s_kern[localidy] = kern[localidy];
    s_img[localidy] = img[globalidx*width + globalidy];
    __syncthreads();

    if (i < height && j < width) {
        float sum = 0;
        for (size_t k = 0; k < kern_w; k++) {
            int offset = k - floorf(kern_w/2);
            p = base + offset;
            if (p / width == base / width) { // same row
                if (p >= 0 && p <= height * width) {
                    if (localidy + offset < blockDim.y)
                        sum += s_img[localidy + offset] * s_kern[k];
                    else
                        sum += img[p] * kern[k];
                }
            }
        }
        out[base] = sum;
    }
}


__global__
void gpu_vconvolve(float *img, float *out, int width, int height, float *kern, int kern_w) {
    extern __shared__ float smem[];
    float *s_kern = smem;
    float *s_img = &smem[kern_w];
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y; 
    int localidx = threadIdx.x;
    int localidy = threadIdx.y;
    int globalidx = localidx + blockIdx.x*blockDim.x;
    int globalidy = localidy + blockIdx.y*blockDim.y;
    int base = i*width + j;
    int p;

    // load to shared mem
    if (localidx < kern_w)
        s_kern[localidx] = kern[localidx];
    s_img[localidx] = img[globalidx*width + globalidy];
    __syncthreads();


    if (i < height && j < width) {
        float sum = 0;
        for (size_t k = 0; k < kern_w; k++) {
            int offset = (k - floorf(kern_w/2)) * width;
            p = base + offset;
            if (p >= 0 && p < width * height) {
                if (localidx + offset < blockDim.x) 
                    sum += s_img[localidx + offset] * s_kern[k];
                else
                    sum += img[p] * s_kern[k];
            }
        }
        out[base] = sum;
    }
}

__global__
void gpu_magdir(float *hori, float *vert, float *mag, float *dir, int height, int width) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = i*width+j;
    float ho = hori[k];
    float ve = vert[k];

    mag[k] = sqrtf((ho * ho) + (ve * ve));
    dir[k] = atan2f(ho, ve);
} 


int main(int argc, char *argv[]) {

    int height;
    int width;
    float sigma;
    int kern_w;
    struct timeval convstart, convstop, magdirstart, magdirstop, htodstart, htodstop, dtohstart, dtohstop, compstart, compstop;

    // host
    float *h_img;
    float *h_mag;
    float *h_dir;
    float *h_vkern;
    float *h_hkern;
    float *h_vderiv;
    float *h_hderiv;

    // device
    float *d_img;
    float *d_temp;
    float *d_hori;
    float *d_vert;
    float *d_mag;
    float *d_dir;
    float *d_vkern;
    float *d_hkern;
    float *d_vderiv;
    float *d_hderiv;
 
    // argparse
    if (argc != 3) {
        fprintf(stderr, "usage: ./canny <image path> <sigma>\n");
        return -1;
    }
    sigma = atof(argv[2]);
    if (sigma <= 0) {
        fprintf(stderr, "invalid sigma: %s\n", argv[2]);
        return -1;
    }

    // cuda setup
    hipSetDevice(GPU_NO);

    // image prep
    read_image_template<float>(argv[1], &h_img, &width, &height);
    h_mag = (float *) calloc(width*height, sizeof(float));
    h_dir = (float *) calloc(width*height, sizeof(float));
    hipMalloc((void **)&d_img, sizeof(float)*width*height);
    hipMalloc((void **)&d_temp, sizeof(float)*width*height);
    hipMalloc((void **)&d_hori, sizeof(float)*width*height);
    hipMalloc((void **)&d_vert, sizeof(float)*width*height);
    hipMalloc((void **)&d_mag, sizeof(float)*width*height);
    hipMalloc((void **)&d_dir, sizeof(float)*width*height);

    // computation start
    gettimeofday(&compstart, NULL);

    // prepare canny kernels
    kern_w = 2 * round(2.5 * sigma - 0.5) + 1;
    h_vkern = (float *) calloc(kern_w, sizeof(float));
    h_hkern = (float *) calloc(kern_w, sizeof(float));
    h_vderiv = (float *) calloc(kern_w, sizeof(float));
    h_hderiv = (float *) calloc(kern_w, sizeof(float));
    hipMalloc((void **)&d_vkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_vderiv, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hderiv, sizeof(float)*kern_w);

    // calculate ckernels
    g_kern(h_vkern, sigma);
    g_kern(h_hkern, sigma);
    g_deriv(h_vderiv, sigma);
    g_deriv(h_hderiv, sigma);

    // transfer ckernels
    gettimeofday(&htodstart, NULL);
    hipMemcpy(d_vkern, h_vkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hkern, h_hkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_vderiv, h_vderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hderiv, h_hderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);

    // transfer image
    hipMemcpy(d_img, h_img, sizeof(float)*width*height, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    gettimeofday(&htodstop, NULL);


    hipDeviceSynchronize();
    gettimeofday(&convstart, NULL);

    // horizontal convolve
    dim3 h_dB(1, conv_size);
    dim3 h_dG(height, width/conv_size);
    int memsize = sizeof(float) * (conv_size + kern_w);
    gpu_hconvolve<<<h_dG,h_dB,memsize>>>(d_img, d_temp, width, height, d_hkern, kern_w);
    gpu_hconvolve<<<h_dG,h_dB,memsize>>>(d_temp, d_hori, width, height, d_hderiv, kern_w);

    // vertical convolve
    dim3 v_dB(conv_size, 1);
    dim3 v_dG(height/conv_size, width);
    gpu_vconvolve<<<v_dG,v_dB,memsize>>>(d_img, d_temp, width, height, d_vkern, kern_w);
    gpu_vconvolve<<<v_dG,v_dB,memsize>>>(d_temp, d_vert, width, height, d_vderiv, kern_w);
    hipDeviceSynchronize();
    gettimeofday(&convstop, NULL);

    // mag & dir
    gettimeofday(&magdirstart, NULL);
    dim3 md_dB(blocksize, blocksize);
    dim3 md_dG(width/blocksize, height/blocksize);
    gpu_magdir<<<md_dG,md_dB>>>(d_hori, d_vert, d_mag, d_dir, height, width);
    hipDeviceSynchronize();
    gettimeofday(&magdirstop, NULL);

    // pull results
    gettimeofday(&dtohstart, NULL);
    hipMemcpy(h_mag, d_mag, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipMemcpy(h_dir, d_dir, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 
    gettimeofday(&dtohstop, NULL);

    // computation end
    gettimeofday(&compstop, NULL);

    // write results
    write_image_template<float>("magnitude.pgm", h_mag, width, height);
    write_image_template<float>("direction.pgm", h_dir, width, height);
    printf("%0.2f,%0.2f,%0.2f,%0.2f,%0.2f\n",
        timecalc(convstart, convstop),
        timecalc(magdirstart, magdirstop),
        timecalc(htodstart, htodstop),
        timecalc(dtohstart, dtohstop),
        timecalc(compstart, compstop)
    );

    // free
    free(h_vkern);
    free(h_hkern);
    free(h_vderiv);
    free(h_hderiv);
    free(h_mag);
    free(h_dir);
    hipFree(d_img);
    hipFree(d_temp);
    hipFree(d_hori);
    hipFree(d_vert);
    hipFree(d_mag);
    hipFree(d_dir);
}


void print_k(float *k, int len) {
    for (size_t i = 0; i < len; i++) {
        printf("[%f]", k[i]);
    }
    printf("\n");
}


float timecalc(struct timeval start, struct timeval end) {
    float ns = (end.tv_sec*1000000 + end.tv_usec) - (start.tv_sec*1000000 + start.tv_usec);
    return ns / 1000.0;
}


void g_kern(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;

    for (size_t i = 0; i < w; i++) {
        k[i] = exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum += k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
}


void g_deriv(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;
    
    for (size_t i = 0; i < w; i++) {
        k[i] = -1.0 * (i-a) * exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum -= i * k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
    // flip
    for (size_t i = 0; i < (w/2); i++) {
        float temp = k[w-1-i];
        k[w-1-i] = k[i];
        k[i] = temp;
    }
}
