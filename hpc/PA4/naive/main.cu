#include "hip/hip_runtime.h"
/*
    Michael Kmak    
    ECPE 251 - High-Performance Computing
    PA4 - Canny Edge Sate 1 - GPU

    usage: ./canny <image path> <sigma>

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

//#include "sort.h"
#include "image_template.h"

#define GPU_NO 1 // 85 % 4
#define BLOCKSIZE 8

void print_k(float *k, int len);
float timecalc(struct timeval start, struct timeval end);
void g_kern(float *k, float sigma);
void g_deriv(float *k, float sigma);


__global__
void gpu_hconvolve(float *img, float *out, int width, int height, float *kern, int kern_w) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y; 
    int base = i*width + j;
    int p;

    if (i < height && j < width) {
        float sum = 0;
        for (size_t k = 0; k < kern_w; k++) {
            int offset = k - floorf(kern_w/2);
            p = base + offset;
            if (p / width == base / width) { // same row
                if (p >= 0 && p <= height * width) {
                    sum += img[p] * kern[k];
                }
            }
        }
        out[base] = sum;
    }
}


__global__
void gpu_vconvolve(float *img, float *out, int width, int height, float *kern, int kern_w) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y; 
    int base = i*width + j;
    int p;

    if (i < height && j < width) {
        float sum = 0;
        for (size_t k = 0; k < kern_w; k++) {
            int offset = (k - floorf(kern_w/2)) * width;
            p = base + offset;
            if (p >= 0 && p < width * height) {
                sum += img[p] * kern[k];
            }
        }
        out[base] = sum;
    }
}

__global__
void gpu_magdir(float *hori, float *vert, float *mag, float *dir, int height, int width) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = i*width+j;
    mag[k] = sqrtf((hori[k] * hori[k]) + (vert[k] * vert[k]));
    dir[k] = atan2f(hori[k], vert[k]);
} 


int main(int argc, char *argv[]) {

    int height;
    int width;
    float sigma;
    int kern_w;
    struct timeval convstart, convstop, magdirstart, magdirstop, htodstart, htodstop, dtohstart, dtohstop, compstart, compstop;

    // host
    float *h_img;
    float *h_mag;
    float *h_dir;
    float *h_vkern;
    float *h_hkern;
    float *h_vderiv;
    float *h_hderiv;

    // device
    float *d_img;
    float *d_temp;
    float *d_hori;
    float *d_vert;
    float *d_mag;
    float *d_dir;
    float *d_vkern;
    float *d_hkern;
    float *d_vderiv;
    float *d_hderiv;
 
    // argparse
    if (argc != 3) {
        fprintf(stderr, "usage: ./canny <image path> <sigma>\n");
        return -1;
    }
    sigma = atof(argv[2]);
    if (sigma <= 0) {
        fprintf(stderr, "invalid sigma: %s\n", argv[2]);
        return -1;
    }

    // cuda setup
    hipSetDevice(GPU_NO);

    // image prep
    read_image_template<float>(argv[1], &h_img, &width, &height);
    h_mag = (float *) calloc(width*height, sizeof(float));
    h_dir = (float *) calloc(width*height, sizeof(float));
    hipMalloc((void **)&d_img, sizeof(float)*width*height);
    hipMalloc((void **)&d_temp, sizeof(float)*width*height);
    hipMalloc((void **)&d_hori, sizeof(float)*width*height);
    hipMalloc((void **)&d_vert, sizeof(float)*width*height);
    hipMalloc((void **)&d_mag, sizeof(float)*width*height);
    hipMalloc((void **)&d_dir, sizeof(float)*width*height);

    // computation start
    gettimeofday(&compstart, NULL);

    // prepare canny kernels
    kern_w = 2 * round(2.5 * sigma - 0.5) + 1;
    h_vkern = (float *) calloc(kern_w, sizeof(float));
    h_hkern = (float *) calloc(kern_w, sizeof(float));
    h_vderiv = (float *) calloc(kern_w, sizeof(float));
    h_hderiv = (float *) calloc(kern_w, sizeof(float));
    hipMalloc((void **)&d_vkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_vderiv, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hderiv, sizeof(float)*kern_w);

    // calculate ckernels
    g_kern(h_vkern, sigma);
    g_kern(h_hkern, sigma);
    g_deriv(h_vderiv, sigma);
    g_deriv(h_hderiv, sigma);

    // transfer ckernels
    gettimeofday(&htodstart, NULL);
    hipMemcpy(d_vkern, h_vkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hkern, h_hkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_vderiv, h_vderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hderiv, h_hderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);

    // transfer image
    hipMemcpy(d_img, h_img, sizeof(float)*width*height, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    gettimeofday(&htodstop, NULL);

    // GPU convolve
    hipDeviceSynchronize();
    gettimeofday(&convstart, NULL);
    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 dimGrid(width/BLOCKSIZE, height/BLOCKSIZE);
    gpu_hconvolve<<<dimGrid,dimBlock>>>(d_img, d_temp, width, height, d_hkern, kern_w);
    gpu_hconvolve<<<dimGrid,dimBlock>>>(d_temp, d_hori, width, height, d_hderiv, kern_w);
    gpu_vconvolve<<<dimGrid,dimBlock>>>(d_img, d_temp, width, height, d_vkern, kern_w);
    gpu_vconvolve<<<dimGrid,dimBlock>>>(d_temp, d_vert, width, height, d_vderiv, kern_w);
    hipDeviceSynchronize();
    gettimeofday(&convstop, NULL);
    gettimeofday(&magdirstart, NULL);
    gpu_magdir<<<dimGrid,dimBlock>>>(d_hori, d_vert, d_mag, d_dir, height, width);
    hipDeviceSynchronize();
    gettimeofday(&magdirstop, NULL);

    // pull results
    gettimeofday(&dtohstart, NULL);
    hipMemcpy(h_mag, d_mag, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipMemcpy(h_dir, d_dir, sizeof(float)*width*height, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 
    gettimeofday(&dtohstop, NULL);

    // computation end
    gettimeofday(&compstop, NULL);

    // write results
    write_image_template<float>("magnitude.pgm", h_mag, width, height);
    write_image_template<float>("direction.pgm", h_dir, width, height);
    printf("%0.2f,%0.2f,%0.2f,%0.2f,%0.2f\n",
        timecalc(convstart, convstop),
        timecalc(magdirstart, magdirstop),
        timecalc(htodstart, htodstop),
        timecalc(dtohstart, dtohstop),
        timecalc(compstart, compstop)
    );

    // free
    free(h_vkern);
    free(h_hkern);
    free(h_vderiv);
    free(h_hderiv);
    free(h_mag);
    free(h_dir);
    hipFree(d_img);
    hipFree(d_temp);
    hipFree(d_hori);
    hipFree(d_vert);
    hipFree(d_mag);
    hipFree(d_dir);
}


void print_k(float *k, int len) {
    for (size_t i = 0; i < len; i++) {
        printf("[%f]", k[i]);
    }
    printf("\n");
}


float timecalc(struct timeval start, struct timeval end) {
    float ns = (end.tv_sec*1000000 + end.tv_usec) - (start.tv_sec*1000000 + start.tv_usec);
    return ns / 1000.0;
}


void g_kern(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;

    for (size_t i = 0; i < w; i++) {
        k[i] = exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum += k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
}


void g_deriv(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;
    
    for (size_t i = 0; i < w; i++) {
        k[i] = -1.0 * (i-a) * exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum -= i * k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
    // flip
    for (size_t i = 0; i < (w/2); i++) {
        float temp = k[w-1-i];
        k[w-1-i] = k[i];
        k[i] = temp;
    }
}
